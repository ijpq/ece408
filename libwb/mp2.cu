#include "hip/hip_runtime.h"
#include    "wb.h"

#define wbCheck(stmt) do {                                 \
        hipError_t err = stmt;                            \
        if (err != hipSuccess) {                          \
            wbLog(ERROR, "Failed to run stmt ", #stmt);    \
            return -1;                                     \
        }                                                  \
    } while(0)

// Compute C = A * B
__global__ void matrixMultiply(float * A, float * B, float * C,
			       int numARows, int numAColumns,
			       int numBRows, int numBColumns,
			       int numCRows, int numCColumns) {
    //@@ Insert code to implement matrix multiplication here
    int tid_x = blockIdx.x * blockDim.x + threadIdx.x;
    int tid_y = blockIdx.y * blockDim.y + threadIdx.y;
    if (tid_x >= numCColumns || tid_y >= numCRows)
        return;
    // numAcol = numBrow
    //int strideB = numBColumns;
    int calcu_size = numBColumns;
    
    float C_value = 0.0;
    for (int i=0;i<calcu_size;i++) {
        C_value += B[tid_y*numBColumns+i] * A[tid_x+i*numAColumns];
    }
    C[tid_y*numCColumns+tid_x] = C_value;
    return ;
}

int main(int argc, char ** argv) {
    wbArg_t args;
    float * hostA; // The A matrix
    float * hostB; // The B matrix
    float * hostC; // The output C matrix
    float * deviceA;
    float * deviceB;
    float * deviceC;
    int numARows; // number of rows in the matrix A
    int numAColumns; // number of columns in the matrix A
    int numBRows; // number of rows in the matrix B
    int numBColumns; // number of columns in the matrix B
    int numCRows; // number of rows in the matrix C (you have to set this)
    int numCColumns; // number of columns in the matrix C (you have to set this)

    args = wbArg_read(argc, argv);

    wbTime_start(Generic, "Importing data and creating memory on host");
    hostA = (float *) wbImport(wbArg_getInputFile(args, 0), &numARows, &numAColumns);
    hostB = (float *) wbImport(wbArg_getInputFile(args, 1), &numBRows, &numBColumns);
    //@@ Set numCRows and numCColumns
    numCRows = numBRows;
    numCColumns = numAColumns;
    //@@ Allocate the hostC matrix
    wbTime_stop(Generic, "Importing data and creating memory on host");

    wbLog(TRACE, "The dimensions of A are ", numARows, " x ", numAColumns);
    wbLog(TRACE, "The dimensions of B are ", numBRows, " x ", numBColumns);
    hostC = (float *)malloc(sizeof(float)*numCRows*numCColumns);

    wbTime_start(GPU, "Allocating GPU memory.");
    //@@ Allocate GPU memory here
    hipMalloc(&deviceA, sizeof(float)*numARows*numAColumns);
    hipMalloc(&deviceB, sizeof(float)*numBRows*numBColumns);
    hipMalloc(&deviceC, sizeof(float)*numCRows*numCColumns);

    wbTime_stop(GPU, "Allocating GPU memory.");

    wbTime_start(GPU, "Copying input memory to the GPU.");
    //@@ Copy memory to the GPU here
    hipMemcpy(deviceA, hostA, sizeof(float)*numARows*numAColumns, hipMemcpyHostToDevice);
    hipMemcpy(deviceB, hostB, sizeof(float)*numBRows*numBColumns, hipMemcpyHostToDevice);
    hipMemcpy(deviceC, hostC, sizeof(float)*numCRows*numCColumns, hipMemcpyHostToDevice);

    wbTime_stop(GPU, "Copying input memory to the GPU.");
    
    //@@ Initialize the grid and block dimensions here
    
    dim3 block(32, 32);
    dim3 grid((numCColumns+block.x-1)/block.x, (numCRows+block.y-1)/block.y);
    wbTime_start(Compute, "Performing CUDA computation");
    //@@ Launch the GPU Kernel here
    matrixMultiply<<<grid,block>>>(deviceA,deviceB,deviceC,numARows,numAColumns,numBRows,numBColumns,numCRows,numCColumns);
    hipDeviceSynchronize();
    wbTime_stop(Compute, "Performing CUDA computation");
    
    wbTime_start(Copy, "Copying output memory to the CPU");
    //@@ Copy the GPU memory back to the CPU here
    hipMemcpy(hostA, deviceA, sizeof(float)*numARows*numAColumns, hipMemcpyDeviceToHost);
    hipMemcpy(hostB, deviceB, sizeof(float)*numBRows*numBColumns, hipMemcpyDeviceToHost);
    hipMemcpy(hostC, deviceC, sizeof(float)*numCRows*numCColumns, hipMemcpyDeviceToHost);

    wbTime_stop(Copy, "Copying output memory to the CPU");

    wbTime_start(GPU, "Freeing GPU Memory");
    //@@ Free the GPU memory here
    hipFree(deviceA);
    hipFree(deviceB);
    hipFree(deviceC);

    wbTime_stop(GPU, "Freeing GPU Memory");

    wbSolution(args, hostC, numCRows, numCColumns);

    free(hostA);
    free(hostB);
    free(hostC);

    return 0;
}

